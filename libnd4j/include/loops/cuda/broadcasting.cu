#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <op_boilerplate.h>
#include <loops/broadcasting.h>
#include <loops/legacy_ops.h>


template <typename T, typename OpClass>
__device__ void broadcastSimpleGeneric(
		T *x,
		Nd4jLong *xShapeInfo,
		T *y,
		Nd4jLong *yShapeInfo,
		T *result,
		Nd4jLong *resultShapeInfo,
		int *dimension,
		int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {


	functions::broadcast::Broadcast<T>::template transformCuda<OpClass>(
			x,
			xShapeInfo,
			y,
			yShapeInfo,
			result,
			resultShapeInfo,
			dimension,
			dimensionLength,
			NULL,
			tadOnlyShapeInfo,
			tadOffsets,
			tadOnlyShapeInfoZ,
			tadOffsetsZ);
}

// broadcast kernel call
DISPATCH_KERNEL_SIMPLE(broadcastSimple_, broadcastSimpleGeneric, float, INPUT(float *x, Nd4jLong *xShapeInfo, float *y, Nd4jLong *yShapeInfo, float *result, Nd4jLong *resultShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ), PARAMS(x, xShapeInfo, y, yShapeInfo, result, resultShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_OPS))
DISPATCH_KERNEL_SIMPLE(broadcastSimple_, broadcastSimpleGeneric, double, INPUT(double *x, Nd4jLong *xShapeInfo, double *y, Nd4jLong *yShapeInfo, double *result, Nd4jLong *resultShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ), PARAMS(x, xShapeInfo, y, yShapeInfo, result, resultShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_OPS))
DISPATCH_KERNEL_SIMPLE(broadcastSimple_, broadcastSimpleGeneric, float16, INPUT(float16 *x, Nd4jLong *xShapeInfo, float16 *y, Nd4jLong *yShapeInfo, float16 *result, Nd4jLong *resultShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ), PARAMS(x, xShapeInfo, y, yShapeInfo, result, resultShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_OPS))


namespace functions {
    namespace broadcast {

        template <>
        __host__ void Broadcast<float>::executeBroadcast(dim3 launchDims, hipStream_t *stream, int opNum, float *x, Nd4jLong *xShapeInfo, float *y, Nd4jLong *yShapeInfo, float *result, Nd4jLong *resultShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {
            DISPATCH_SIMPLE(broadcastSimple, float, PARAMS(x, xShapeInfo, y, yShapeInfo, result, resultShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_OPS))

	        DEBUG_KERNEL(stream, opNum);
        }

        template <>
        __host__ void Broadcast<float16>::executeBroadcast(dim3 launchDims, hipStream_t *stream, int opNum, float16 *x, Nd4jLong *xShapeInfo, float16 *y, Nd4jLong *yShapeInfo, float16 *result, Nd4jLong *resultShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {
            DISPATCH_SIMPLE(broadcastSimple, float16, PARAMS(x, xShapeInfo, y, yShapeInfo, result, resultShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        __host__ void Broadcast<double>::executeBroadcast(dim3 launchDims, hipStream_t *stream, int opNum, double *x, Nd4jLong *xShapeInfo, double *y, Nd4jLong *yShapeInfo, double *result, Nd4jLong *resultShapeInfo, int *dimension, int dimensionLength, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {
            DISPATCH_SIMPLE(broadcastSimple, double, PARAMS(x, xShapeInfo, y, yShapeInfo, result, resultShapeInfo, dimension, dimensionLength, tadOnlyShapeInfo, tadOffsets, tadOnlyShapeInfoZ, tadOffsetsZ), OPS_A(BROADCAST_OPS))

            DEBUG_KERNEL(stream, opNum);
        }


        template <typename T>
        template <typename OpType>
		__device__ void Broadcast<T>::transformCuda(
		T *x,
		Nd4jLong *xShapeInfo,
		T *y,
		Nd4jLong *yShapeInfo,
		T *result,
		Nd4jLong *resultShapeInfo,
		int *dimension,
		int dimensionLength, UnifiedSharedMemory *manager, Nd4jLong *tadOnlyShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadOnlyShapeInfoZ, Nd4jLong *tadOffsetsZ) {

		//decompose in to several sub tads after
		//moving all dimensions (in sorted order)
		//to the back.
		//permuted version of the x shape info for setting up the tad problem
	  __shared__ Nd4jLong tadLength;
      __shared__ Nd4jLong tadEWS;
      __shared__ int tadRank;
      __shared__ int numTads;
      __shared__ Nd4jLong *tadShape;
      __shared__ Nd4jLong *tadStride;
      __shared__ Nd4jLong yEWS;
      __shared__ Nd4jLong zEWS;
      __shared__ int zRank;
      __shared__ Nd4jLong *zShape;
      __shared__ Nd4jLong *zStride;
      __shared__ int yRank;
      __shared__ Nd4jLong *yShape;
      __shared__ Nd4jLong *yStride;
      if (threadIdx.x == 0) {
        if (tadOnlyShapeInfoZ == nullptr) {
            tadOnlyShapeInfoZ = tadOnlyShapeInfo;
            tadOffsetsZ = tadOffsets;
        }

   	    tadLength = shape::tadLength(xShapeInfo, dimension, dimensionLength);
        tadEWS = shape::elementWiseStride(tadOnlyShapeInfo);
        numTads = shape::length(xShapeInfo) / tadLength;
        yEWS = shape::elementWiseStride(yShapeInfo);
      	zEWS = shape::elementWiseStride(tadOnlyShapeInfoZ);

        if (tadEWS < 1 || zEWS < 1 || yEWS < 1 || dimensionLength > 1) {
            tadRank = shape::rank(tadOnlyShapeInfo);
            tadShape = shape::shapeOf(tadOnlyShapeInfo);
      	    tadStride = shape::stride(tadOnlyShapeInfo);
      	    zRank = shape::rank(tadOnlyShapeInfoZ);
      	    zShape = shape::shapeOf(tadOnlyShapeInfoZ);
      	    zStride = shape::stride(tadOnlyShapeInfoZ);
      	    yRank = shape::rank(yShapeInfo);
      	    yShape = shape::shapeOf(yShapeInfo);
      	    yStride = shape::stride(yShapeInfo);
        }
      }
      __syncthreads();

		for (int r = blockIdx.x; r < numTads; r += gridDim.x) {


            __shared__ Nd4jLong tadOffsetForBlock;
            __shared__ Nd4jLong tadOffsetForBlockZ;
            __shared__ T *rR;
            __shared__ T *rX;
            if (threadIdx.x == 0) {
                tadOffsetForBlockZ = tadOffsetsZ[r];
                if (result != x)
                    tadOffsetForBlock = tadOffsets[r];
                else
                    tadOffsetForBlock = tadOffsetForBlockZ;

                rR = result + tadOffsetForBlockZ;
                rX = x + tadOffsetForBlock;
            }
            __syncthreads();


            if(tadEWS > 0 && zEWS > 0 && yEWS > 0 && dimensionLength == 1) {
            	if (tadEWS == 1 && yEWS == 1 && zEWS == 1) {
                	for (int i = threadIdx.x; i < tadLength; i+= blockDim.x) {
                    	rR[i] = OpType::op(rX[i], y[i]);
                	}
                } else {
					for (int i = threadIdx.x; i < tadLength; i+= blockDim.x) {
                    	rR[i * zEWS] = OpType::op(rX[i * tadEWS], y[i * yEWS]);
                	}
                }
            }
            else {
                Nd4jLong xCoord[MAX_RANK];
                Nd4jLong yCoord[MAX_RANK];
                Nd4jLong zCoord[MAX_RANK];

                for (Nd4jLong i = threadIdx.x; i < tadLength; i+= blockDim.x) {

                    if (shape::order(tadOnlyShapeInfo) == 'c') {
                        shape::ind2subC(tadRank,tadShape, i, tadLength, xCoord);
                        shape::ind2subC(yRank, yShape, i, tadLength, yCoord);
                    } else {
                        shape::ind2sub(tadRank,tadShape, i, tadLength, xCoord);
                        shape::ind2sub(yRank, yShape, i, tadLength, yCoord);
                    }

                    if (shape::order(tadOnlyShapeInfoZ) == 'c')
                        shape::ind2subC(zRank,zShape, i, tadLength, zCoord);
                    else
                        shape::ind2sub(zRank,zShape, i, tadLength, zCoord);

                    auto xOffset = shape::getOffset(tadOffsetForBlock, tadShape, tadStride, xCoord, tadRank);
                    auto zOffset = shape::getOffset(tadOffsetForBlockZ, zShape, zStride, zCoord, zRank);
                    auto yOffset = shape::getOffset(0, yShape, yStride, yCoord, yRank);
                    result[zOffset] = OpType::op(x[xOffset], y[yOffset]);
                }
            }
		}
	}
    }
}