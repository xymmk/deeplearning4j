#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#ifndef SCALAR_CU
#define SCALAR_CU

#include "../scalar.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <op_boilerplate.h>
#include <helpers/TAD.h>
#include <types/float16.h>



template <typename T, typename OpType>
__device__ void scalarAlongDimensionGeneric(T *x,
                                            Nd4jLong *xShapeInfo,
                                            T *extraParams,
                                            T *z,
                                            Nd4jLong *zShapeInfo,
                                            T *scalars,
                                            int *dimension,
                                            int dimensionLength,
                                            Nd4jLong *tadShapeInfo,
                                            Nd4jLong *tadOffsets,
                                            Nd4jLong *tadShapeInfoZ,
                                            Nd4jLong *tadOffsetsZ) {

    functions::scalar::ScalarTransform<T>::template transformCuda<OpType>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}

template <typename T, typename OpClass>
__device__ void scalarSimpleGeneric(
        Nd4jLong n,
        T dx,
        T *dy,
        Nd4jLong incy, T *params,
        T *result, Nd4jLong resultStride, int *allocationBuffer) {

    functions::scalar::ScalarTransform<T>::template transformCuda<OpClass>(
            n,
            dx,
            dy,
            incy,
            params,
            result,
            resultStride,
            allocationBuffer,
            NULL);
}

/*
// LEGACY KERNELS,
template <typename T>
__device__ void scalarGenericIndexes(
        int opNum,
        Nd4jLong n,
        T dx,
        T *dy,
        T *params,
        T *result,int *indexes, int *allocationBuffer) {

    __shared__ UnifiedSharedMemory *manager;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        manager = new(shmem) UnifiedSharedMemory((int *) shmem);
        manager->init(sizeof(UnifiedSharedMemory), 0, sizeof(functions::scalar::ScalarTransform<T>), sizeof(shape::TAD), 0);
    }
    __syncthreads();

    functions::scalar::ScalarTransform<T>::transform(
            opNum,
            n,
            dx,
            dy,
            params,
            result,
            indexes,
            allocationBuffer,
            manager);
}

__global__ void scalarDoubleIndexes(
        int opNum,
        Nd4jLong n,
        double dx,
        double *dy,
        double *params,
        double *result,int *indexes, int *allocationBuffer) {
    scalarGenericIndexes<double>(opNum,
                                 n,
                                 dx,
                                 dy,
                                 params,
                                 result,
                                 indexes, allocationBuffer);
}

__global__ void scalarFloatIndexes(
        int opNum,
        Nd4jLong n,
        float dx,
        float *dy,
        float *params,
        float *result,
        int *indexes, int *allocationBuffer) {
    scalarGenericIndexes<float>(opNum,
                                n,
                                dx,
                                dy,
                                params,
                                result,
                                indexes, allocationBuffer);
}
*/

template <typename T, typename OpClass>
__device__ void scalarSimpleGeneric(
        T dx,
        T *dy,
        Nd4jLong *xShapeInfo,
        T *params,
        T *result,
        Nd4jLong *resultShapeInfo,
        int *allocationBuffer) {

    functions::scalar::ScalarTransform<T>::template transformCuda<OpClass>(
            dx,
            dy,
            xShapeInfo,
            params,
            result,
            resultShapeInfo,
            allocationBuffer,
            nullptr);
}



// ScalarOp Along Dimension kernels
DISPATCH_KERNEL_SIMPLE(scalarAlongDimension_, scalarAlongDimensionGeneric, float, INPUT(float *x, Nd4jLong *xShapeInfo, float *extraParams, float *z, Nd4jLong *zShapeInfo, float *scalars, int *dimension, int dimensionLength, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ), PARAMS(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), OPS_A(SCALAR_OPS))
DISPATCH_KERNEL_SIMPLE(scalarAlongDimension_, scalarAlongDimensionGeneric, double, INPUT(double *x, Nd4jLong *xShapeInfo, double *extraParams, double *z, Nd4jLong *zShapeInfo, double *scalars, int *dimension, int dimensionLength, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ), PARAMS(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), OPS_A(SCALAR_OPS))
DISPATCH_KERNEL_SIMPLE(scalarAlongDimension_, scalarAlongDimensionGeneric, float16, INPUT(float16 *x, Nd4jLong *xShapeInfo, float16 *extraParams, float16 *z, Nd4jLong *zShapeInfo, float16 *scalars, int *dimension, int dimensionLength, Nd4jLong *tadShapeInfo, Nd4jLong *tadOffsets, Nd4jLong *tadShapeInfoZ, Nd4jLong *tadOffsetsZ), PARAMS(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), OPS_A(SCALAR_OPS))

// scalar shape
DISPATCH_KERNEL_SIMPLE(scalarSimpleShaped_, scalarSimpleGeneric, float, INPUT(float dx, float *dy, Nd4jLong *xShapeInfo, float *params, float *result, Nd4jLong *resultShapeInfo, int *allocationBuffer), PARAMS(dx, dy, xShapeInfo, params, result, resultShapeInfo, allocationBuffer), OPS_A(SCALAR_OPS))
DISPATCH_KERNEL_SIMPLE(scalarSimpleShaped_, scalarSimpleGeneric, double, INPUT(double dx, double *dy, Nd4jLong *xShapeInfo, double *params, double *result, Nd4jLong *resultShapeInfo, int *allocationBuffer), PARAMS(dx, dy, xShapeInfo, params, result, resultShapeInfo, allocationBuffer), OPS_A(SCALAR_OPS))
DISPATCH_KERNEL_SIMPLE(scalarSimpleShaped_, scalarSimpleGeneric, float16, INPUT(float16 dx, float16 *dy, Nd4jLong *xShapeInfo, float16 *params, float16 *result, Nd4jLong *resultShapeInfo, int *allocationBuffer), PARAMS(dx, dy, xShapeInfo, params, result, resultShapeInfo, allocationBuffer), OPS_A(SCALAR_OPS))

// scalar strided
DISPATCH_KERNEL_SIMPLE(scalarSimpleStrided_, scalarSimpleGeneric, float, INPUT(Nd4jLong n, float dx, float *dy, Nd4jLong incy, float *params, float *result,Nd4jLong resultStride, int *allocationBuffer), PARAMS(n, dx, dy, incy, params, result, resultStride, allocationBuffer), OPS_A(SCALAR_OPS))
DISPATCH_KERNEL_SIMPLE(scalarSimpleStrided_, scalarSimpleGeneric, double, INPUT(Nd4jLong n, double dx, double *dy, Nd4jLong incy, double *params, double *result,Nd4jLong resultStride, int *allocationBuffer), PARAMS(n, dx, dy, incy, params, result, resultStride, allocationBuffer), OPS_A(SCALAR_OPS))
DISPATCH_KERNEL_SIMPLE(scalarSimpleStrided_, scalarSimpleGeneric, float16, INPUT(Nd4jLong n, float16 dx, float16 *dy, Nd4jLong incy, float16 *params, float16 *result,Nd4jLong resultStride, int *allocationBuffer), PARAMS(n, dx, dy, incy, params, result, resultStride, allocationBuffer), OPS_A(SCALAR_OPS))


namespace functions {
    namespace scalar {

    template<>
    void ScalarTransform<float>::executeCudaStrided(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, float *x, Nd4jLong xStride, float *result, Nd4jLong resultStride, float scalar, float *extraParams, Nd4jLong n) {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		    printf("F13 opNum:[%i]\n", opNum);

		int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	    // this macro builds bunch of IF/ELSE selectors for kernel launch
        DISPATCH_SIMPLE(scalarSimpleStrided, float, PARAMS(n, scalar, x, xStride, extraParams, result, resultStride, allocPointer), OPS_A(SCALAR_OPS))
    }


    template<>
    void ScalarTransform<float16>::executeCudaStrided(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, float16 *x, Nd4jLong xStride, float16 *result, Nd4jLong resultStride, float16 scalar, float16 *extraParams, Nd4jLong n) {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		    printf("H13 opNum:[%i]\n", opNum);

		int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	    // this macro builds bunch of IF/ELSE selectors for kernel launch
        DISPATCH_SIMPLE(scalarSimpleStrided, float16, PARAMS(n, scalar, x, xStride, extraParams, result, resultStride, allocPointer), OPS_A(SCALAR_OPS))
    }


    template<>
    void ScalarTransform<double>::executeCudaStrided(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, double *x, Nd4jLong xStride, double *result, Nd4jLong resultStride, double scalar, double *extraParams, Nd4jLong n) {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	    if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		    printf("D13 opNum:[%i]\n", opNum);

		int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	    // this macro builds bunch of IF/ELSE selectors for kernel launch
        DISPATCH_SIMPLE(scalarSimpleStrided, double, PARAMS(n, scalar, x, xStride, extraParams, result, resultStride, allocPointer), OPS_A(SCALAR_OPS))
    }


    template<>
    void ScalarTransform<float16>::executeCudaShaped(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, float16 *x, Nd4jLong *xShapeInfo, float16 *result, Nd4jLong *resultShapeInfo, float16 scalar, float16 *extraParams) {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

        if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		    printf("H14 opNum:[%i]\n", opNum);

		int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

        DISPATCH_SIMPLE(scalarSimpleShaped, float16, PARAMS(scalar, x, xShapeInfo, extraParams, result, resultShapeInfo, allocPointer), OPS_A(SCALAR_OPS))
    }

    template<>
    void ScalarTransform<float>::executeCudaShaped(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, float *x, Nd4jLong *xShapeInfo, float *result, Nd4jLong *resultShapeInfo, float scalar, float *extraParams) {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

        if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		    printf("F14 opNum:[%i]\n", opNum);

        int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

        DISPATCH_SIMPLE(scalarSimpleShaped, float, PARAMS(scalar, x, xShapeInfo, extraParams, result, resultShapeInfo, allocPointer), OPS_A(SCALAR_OPS))
    }

    template<>
    void ScalarTransform<double>::executeCudaShaped(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, double *x, Nd4jLong *xShapeInfo, double *result, Nd4jLong *resultShapeInfo, double scalar, double *extraParams) {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

        if (nd4j::Environment::getInstance()->isDebugAndVerbose())
		    printf("D14 opNum:[%i]\n", opNum);

		int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

        DISPATCH_SIMPLE(scalarSimpleShaped, double, PARAMS(scalar, x, xShapeInfo, extraParams, result, resultShapeInfo, allocPointer), OPS_A(SCALAR_OPS))
    }

    template<>
    void ScalarTransform<double>::executeCudaAlongDimension(dim3& launchDims, Nd4jPointer *extraPointers,int opNum, double *x, Nd4jLong *xShapeInfo, double *z, Nd4jLong *zShapeInfo, double *scalars, double *extraParams, int *dimension, int dimensionLength) {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

        auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
        auto tadShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[12]);
        auto tadOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[13]);

        DISPATCH_SIMPLE(scalarAlongDimension, double, PARAMS(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), OPS_A(SCALAR_OPS))
    }

    template<>
    void ScalarTransform<float>::executeCudaAlongDimension(dim3& launchDims, Nd4jPointer *extraPointers,int opNum, float *x, Nd4jLong *xShapeInfo, float *z, Nd4jLong *zShapeInfo, float *scalars, float *extraParams, int *dimension, int dimensionLength) {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

        auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
        auto tadShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[12]);
        auto tadOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[13]);

        DISPATCH_SIMPLE(scalarAlongDimension, float, PARAMS(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), OPS_A(SCALAR_OPS))
    }

    template<>
    void ScalarTransform<float16>::executeCudaAlongDimension(dim3& launchDims, Nd4jPointer *extraPointers,int opNum, float16 *x, Nd4jLong *xShapeInfo, float16 *z, Nd4jLong *zShapeInfo, float16 *scalars, float16 *extraParams, int *dimension, int dimensionLength) {
        hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

        auto tadShapeInfo = reinterpret_cast<Nd4jLong *>(extraPointers[10]);
        auto tadOffsets = reinterpret_cast<Nd4jLong *>(extraPointers[11]);
        auto tadShapeInfoZ = reinterpret_cast<Nd4jLong *>(extraPointers[12]);
        auto tadOffsetsZ = reinterpret_cast<Nd4jLong *>(extraPointers[13]);

        DISPATCH_SIMPLE(scalarAlongDimension, float16, PARAMS(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), OPS_A(SCALAR_OPS))
    }


        /**
     * Cuda implementation of transform
     * @param dx
     * @param xShapeInfo
     * @param result
     * @param resultShapeInfo
     * @param extraParams
     * @param n
     */
    template<typename T>
    template<typename OpType>
    __device__ void ScalarTransform<T>::transform(
            Nd4jLong n,
            T scalar,
            T *dy,
            T *params,
            T *result,
            Nd4jLong *indexes,
            int *allocationBuffer,
            UnifiedSharedMemory *manager) {
        int totalThreads = gridDim.x * blockDim.x;
        int tid = threadIdx.x;
        Nd4jLong i = blockIdx.x * blockDim.x + tid;

        /* equal, positive, non-unit increments. */
        for (; i < n; i+= totalThreads) {
            result[indexes[i]] = OpType::op(dy[indexes[i]],scalar, params);
        }
    }


    /**
     * Cuda implementation of transform
     * @param dx
     * @param xShapeInfo
     * @param result
     * @param resultShapeInfo
     * @param extraParams
     * @param n
     */
     template<typename T>
     template<typename OpType>
     __device__ void ScalarTransform<T>::transformCuda(
            T scalar,
            T *dy,
            Nd4jLong *shapeInfo,
            T *params,
            T *result,
            Nd4jLong *resultShapeInfo,
            int *allocationBuffer,
            UnifiedSharedMemory *manager) {

        auto xShape = shape::shapeOf(shapeInfo);
        auto xStride = shape::stride(shapeInfo);
        auto xOrder = shape::order(shapeInfo);
        auto xRank = shape::rank(shapeInfo);

        auto xElementWiseStride = shape::elementWiseStride(shapeInfo);
        auto resultElementWiseStride = shape::elementWiseStride(resultShapeInfo);
        auto zShape = shape::shapeOf(resultShapeInfo);
        auto zStride = shape::stride(resultShapeInfo);
        int zRank = shape::rank(resultShapeInfo);

        int totalThreads = gridDim.x * blockDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        __shared__ int length;
        if(threadIdx.x == 0)
            length = shape::length(shapeInfo);
        __syncthreads();


        if(xElementWiseStride >= 1 && resultElementWiseStride >= 1 && xOrder == shape::order(resultShapeInfo)) {
            transformCuda<OpType>(
                    length,
                    scalar,
                    dy,
                    xElementWiseStride,
                    params,
                    result,resultElementWiseStride, allocationBuffer, manager);
        }
        else {
            Nd4jLong xIdx[MAX_RANK];

            for (Nd4jLong i = tid; i < length; i+= totalThreads) {
                shape::ind2sub(xRank, xShape, i, length, xIdx);
                auto  xOffset2 = shape::getOffset(0, xShape, xStride, xIdx, xRank);
                auto resultOffset = shape::getOffset(0, zShape, zStride, xIdx, zRank);

                result[resultOffset] = OpType::op(dy[xOffset2],scalar, params);
            }
        }
    }
/**
  * ScalarOp along dimension
**/
    template<typename T>
    template<typename OpType>
    void __device__ ScalarTransform<T>::transformCuda(T *x,
                                Nd4jLong *xShapeInfo,
                                  T *extraParams,
                                  T *z,
                                  Nd4jLong *zShapeInfo,
                                  T *scalars,
                                  int *dimension,
                                  int dimensionLength,
                                  Nd4jLong *tadShapeInfo,
                                  Nd4jLong *tadOffsets,
                                  Nd4jLong *tadShapeInfoZ,
                                  Nd4jLong *tadOffsetsZ) {


                if (tadShapeInfoZ == nullptr) {
                    tadShapeInfoZ = tadShapeInfo;
                    tadOffsetsZ = tadOffsets;
                }

                // tad preparation
                auto tadEWS = shape::elementWiseStride(tadShapeInfo);
                auto zEWS = shape::elementWiseStride(tadShapeInfo);
                auto tadRank = shape::rank(tadShapeInfo);
                auto tadLength = shape::tadLength(xShapeInfo, dimension, dimensionLength);
                auto numTads =shape::length(xShapeInfo) / tadLength;

                // main loop, rolling over tads
                for (int r = blockIdx.x; r < numTads; r+=gridDim.x) {
                    auto offset = tadOffsets[r];
                    auto offsetZ = tadOffsetsZ[r];
                    T scalar = scalars[r];

                    if (tadEWS >= 1 && zEWS >= 1) {
                        T *oZ = z + offsetZ;
                        T *oX = x + offset;

                       for (int f = threadIdx.x; f < tadLength; f+= blockDim.x) {
                            oZ[f] = OpType::op(oX[f], scalar, extraParams);
                        }
                    } else {
                        // ind2sub loop
                        printf("Super-bad loop visited. Shouldn't ever happen\n");
                    }
                }
    }
    /**
     *
     * @param n
     * @param idx
     * @param dx
     * @param dy
     * @param incy
     * @param params
     * @param result
     * @param blockSize
     */
        template<typename T>
        template<typename OpType>
        __device__ void ScalarTransform<T>::transformCuda(
            Nd4jLong n,
            T dx,
            T *dy,
            Nd4jLong incy,
            T *params,
            T *result,
            Nd4jLong resultStride,
            int *allocationBuffer,
            UnifiedSharedMemory *manager) {

        int totalThreads = gridDim.x * blockDim.x;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        Nd4jLong i = tid;
        if(incy == 1 && resultStride == 1) {
            for (; i < n; i += totalThreads) {
                result[i] = OpType::op(dy[i],dx, params);
            }
        }
        else {
            for (; i < n; i += totalThreads) {
                result[i * resultStride] = OpType::op(dy[i * incy],dx, params);
            }
        }
    }

/*
        static inline __device__ void transformCuda(
            const int opNum,
            T scalar,
            T *dy,
            int *shapeInfo,
            T *params,
            T *result,
            int *resultShapeInfo,
            int *allocationBuffer,
            UnifiedSharedMemory *manager) {
                    DISPATCH_BY_OPNUM(transformCuda, PARAMS(scalar, dy, shapeInfo, params, result, resultShapeInfo, allocationBuffer, manager), SCALAR_OPS);
                    }


        static inline __device__ void transform(
            const int opNum,
            Nd4jLong n,
            T scalar,
            T *dy,
            T *params,
            T *result,
            int *indexes,
            int *allocationBuffer,
            UnifiedSharedMemory *manager) {
                    DISPATCH_BY_OPNUM(transform, PARAMS(n, scalar, dy, params, result, indexes, allocationBuffer, manager), SCALAR_OPS);
        }


        static inline __device__ void transformCuda(
            const int opNum,
            Nd4jLong n,
            T dx,
            T *dy,
            int incy,
            T *params,
            T *result,
            int resultStride,
            int *allocationBuffer,
            UnifiedSharedMemory *manager) {
                    DISPATCH_BY_OPNUM(transformCuda, PARAMS(n, dx, dy, incy, params, result, resultStride, allocationBuffer, manager), SCALAR_OPS);
        }
        */

        BUILD_CALL_1(template __device__ void ScalarTransform<float>::transformCuda, float, (float, float*, Nd4jLong *, float*, float*, Nd4jLong*, int*, UnifiedSharedMemory *), SCALAR_OPS)
        BUILD_CALL_1(template __device__ void ScalarTransform<float16>::transformCuda, float16, (float16, float16*, Nd4jLong *, float16*, float16*, Nd4jLong*, int*, UnifiedSharedMemory *), SCALAR_OPS)
        BUILD_CALL_1(template __device__ void ScalarTransform<double>::transformCuda, double, (double, double*, Nd4jLong *, double*, double*, Nd4jLong*, int*, UnifiedSharedMemory *), SCALAR_OPS)
    }
}



#endif // SCALAR_CU