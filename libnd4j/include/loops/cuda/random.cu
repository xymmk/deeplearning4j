#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <op_boilerplate.h>
#include <loops/random.h>
#include <dll.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>
#include <helpers/DebugHelper.h>

template <typename T, typename OpClass>
static inline __device__ void randomSingleGeneric(
        Nd4jPointer state,
        T *z,
        Nd4jLong *zShapeBuffer,
        T *extraArguments) {


    functions::random::RandomFunction<T>::template execTransformCuda<OpClass>(
            state,
            z,
            zShapeBuffer,
            extraArguments);
}

template <typename T, typename OpClass>
static inline __device__ void randomDoubleGeneric(
        Nd4jPointer state,
        T *x,
        Nd4jLong *xShapeBuffer,
        T *z,
        Nd4jLong *zShapeBuffer,
        T *extraArguments) {


    functions::random::RandomFunction<T>::template execTransformCuda<OpClass>(
            state,
            x,
            xShapeBuffer,
            z,
            zShapeBuffer,
            extraArguments);
}


template <typename T, typename OpClass>
static inline __device__ void randomTripleGeneric(
        Nd4jPointer state,
        T *x,
        Nd4jLong *xShapeBuffer,
        T *y,
        Nd4jLong *yShapeBuffer,
        T *z,
        Nd4jLong *zShapeBuffer,
        T *extraArguments) {


    functions::random::RandomFunction<T>::template execTransformCuda<OpClass>(
            state,
            x,
            xShapeBuffer,
            y,
            yShapeBuffer,
            z,
            zShapeBuffer,
            extraArguments);
}


#ifndef __CLION_IDE__
// here we generate kernels for target operations
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, float, INPUT(Nd4jPointer state, float *z, Nd4jLong *zShapeBuffer, float *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, double, INPUT(Nd4jPointer state, double *z, Nd4jLong *zShapeBuffer, double *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomSingle_, randomSingleGeneric, float16, INPUT(Nd4jPointer state, float16 *z, Nd4jLong *zShapeBuffer, float16 *extraArguments), PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, float, INPUT(Nd4jPointer state, float *x, Nd4jLong *xShapeBuffer, float *z, Nd4jLong *zShapeBuffer, float *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, double, INPUT(Nd4jPointer state, double *x, Nd4jLong *xShapeBuffer, double *z, Nd4jLong *zShapeBuffer, double *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomDouble_, randomDoubleGeneric, float16, INPUT(Nd4jPointer state, float16 *x, Nd4jLong *xShapeBuffer, float16 *z, Nd4jLong *zShapeBuffer, float16 *extraArguments), PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, float, INPUT(Nd4jPointer state, float *x, Nd4jLong *xShapeBuffer, float *y, Nd4jLong *yShapeBuffer, float *z, Nd4jLong *zShapeBuffer, float *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, double, INPUT(Nd4jPointer state, double *x, Nd4jLong *xShapeBuffer, double *y, Nd4jLong *yShapeBuffer, double *z, Nd4jLong *zShapeBuffer, double *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))
DISPATCH_KERNEL_SIMPLE(randomTriple_, randomTripleGeneric, float16, INPUT(Nd4jPointer state, float16 *x, Nd4jLong *xShapeBuffer, float16 *y, Nd4jLong *yShapeBuffer, float16 *z, Nd4jLong *zShapeBuffer, float16 *extraArguments), PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

#endif

namespace functions {
    namespace random {
            template<typename T>
            template<typename OpClass>
            void _CUDA_D RandomFunction<T>::execTransformCuda(Nd4jPointer state, T *x, Nd4jLong *xShapeBuffer, T *y, Nd4jLong *yShapeBuffer, T *z, Nd4jLong *zShapeBuffer, T *extraArguments) {
                if (OpClass::requiresSpecial) {
                    OpClass::specialOpCuda(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments);
                    return;
                } else {

                __shared__ Nd4jLong length;
                __shared__ int xEWS;
                __shared__ int yEWS;
                __shared__ int zEWS;

                __shared__ nd4j::random::RandomBuffer *buffer;
                __shared__ unsigned char *cB;
                __shared__ unsigned char *dB;
                __shared__ nd4j::random::RandomBuffer *devBuffer;
                if (threadIdx.x == 0) {
                    length = shape::length(zShapeBuffer);
                    xEWS = shape::elementWiseStride(xShapeBuffer);
                    yEWS = shape::elementWiseStride(yShapeBuffer);
                    zEWS = shape::elementWiseStride(zShapeBuffer);

                    extern __shared__ unsigned char shmem[];
                    buffer = (nd4j::random::RandomBuffer *) shmem;
                    cB = shmem;
                    devBuffer = reinterpret_cast<nd4j::random::RandomBuffer *> (state);
                    dB = reinterpret_cast<unsigned char *> (state);
                }
                __syncthreads();

                // using this loop instead of memcpy
                for (int e = threadIdx.x; e < sizeof(nd4j::random::RandomBuffer); e+= blockDim.x) {
                    cB[e] = dB[e];
                }
                __syncthreads();


                int tid = blockIdx.x * blockDim.x + threadIdx.x;

                if (xEWS >= 1 && yEWS >= 1 && zEWS >= 1) {
                    for (Nd4jLong e = tid; e < length; e += blockDim.x * gridDim.x) {
                        z[e * zEWS] = OpClass::op(x[e * xEWS], y[e * yEWS], e, length, buffer, extraArguments);
                    }
                } else {
                    // negative ews
                    Nd4jLong xCoord[MAX_RANK];
                    Nd4jLong yCoord[MAX_RANK];
                    Nd4jLong zCoord[MAX_RANK];

                    int xRank = shape::rank(xShapeBuffer);
                    int yRank = shape::rank(yShapeBuffer);
                    int zRank = shape::rank(zShapeBuffer);

                    auto xShape = shape::shapeOf(xShapeBuffer);
                    auto yShape = shape::shapeOf(yShapeBuffer);
                    auto zShape = shape::shapeOf(zShapeBuffer);

                    auto xStride = shape::stride(xShapeBuffer);
                    auto yStride = shape::stride(yShapeBuffer);
                    auto zStride = shape::stride(zShapeBuffer);

                    for (Nd4jLong i = tid; i < length; i += blockDim.x * gridDim.x) {
                        shape::ind2sub(xRank, xShape, i, length, xCoord);
                        shape::ind2sub(yRank, yShape, i, length, yCoord);
                        shape::ind2sub(zRank, zShape, i, length, zCoord);

                        auto xOffset2 = shape::getOffset(0, xShape, xStride, xCoord, xRank);
                        auto yOffset2 = shape::getOffset(0, yShape, yStride, yCoord, yRank);
                        auto zOffset2 = shape::getOffset(0, zShape, zStride, zCoord, zRank);

                        z[zOffset2] = OpClass::op(x[xOffset2], y[yOffset2], i, length, buffer, extraArguments);
                    }
                }

                __syncthreads();
                devBuffer->rewind(length);
                }
            };


            template<typename T>
            template<typename OpClass>
            void _CUDA_D RandomFunction<T>::execTransformCuda(Nd4jPointer state, T *x, Nd4jLong *xShapeBuffer, T *z, Nd4jLong *zShapeBuffer, T *extraArguments) {
                __shared__ Nd4jLong length;
                __shared__ int xEWS;
                __shared__ int zEWS;

                __shared__ nd4j::random::RandomBuffer *buffer;
                __shared__ unsigned char *cB;
                __shared__ unsigned char *dB;
                __shared__ nd4j::random::RandomBuffer *devBuffer;
                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    buffer = (nd4j::random::RandomBuffer *) shmem;
                    cB = shmem;
                    devBuffer = reinterpret_cast<nd4j::random::RandomBuffer *> (state);
                    dB = reinterpret_cast<unsigned char *> (state);

                    length = shape::length(zShapeBuffer);
                    xEWS = shape::elementWiseStride(xShapeBuffer);
                    zEWS = shape::elementWiseStride(zShapeBuffer);
                }
                __syncthreads();

                // using this loop instead of memcpy
                for (int e = threadIdx.x; e < sizeof(nd4j::random::RandomBuffer); e+= blockDim.x) {
                    cB[e] = dB[e];
                }
                __syncthreads();


                if (xEWS >= 1 && zEWS >= 1) {
                    for (Nd4jLong e = blockIdx.x * blockDim.x + threadIdx.x; e < length; e += blockDim.x * gridDim.x) {
                        z[e * zEWS] = OpClass::op(x[e * xEWS], e, length, buffer, extraArguments);
                    }
                } else {
                    // ind2sub branch
                    Nd4jLong xCoord[MAX_RANK];
                    Nd4jLong zCoord[MAX_RANK];

                    int xRank = shape::rank(xShapeBuffer);
                    int zRank = shape::rank(zShapeBuffer);

                    auto xShape = shape::shapeOf(xShapeBuffer);
                    auto zShape = shape::shapeOf(zShapeBuffer);

                    auto xStride = shape::stride(xShapeBuffer);
                    auto zStride = shape::stride(zShapeBuffer);

                    for (Nd4jLong i = blockIdx.x * blockDim.x + threadIdx.x; i < length; i += blockDim.x * gridDim.x) {
                        shape::ind2sub(xRank, xShape, i, length, xCoord);
                        shape::ind2sub(zRank, zShape, i, length, zCoord);

                        auto xOffset2 = shape::getOffset(0, xShape, xStride, xCoord, xRank);
                        auto zOffset2 = shape::getOffset(0, zShape, zStride, zCoord, zRank);

                        z[zOffset2] = OpClass::op(x[xOffset2], i, length, buffer, extraArguments);
                    }
                }

                __syncthreads();
                devBuffer->rewind(length);
            }


            template<typename T>
            template<typename OpClass>
            void _CUDA_D RandomFunction<T>::execTransformCuda(Nd4jPointer state, T *z, Nd4jLong *zShapeBuffer, T *extraArguments) {
                Nd4jLong length = shape::length(zShapeBuffer);
                int ews = shape::elementWiseStride(zShapeBuffer);

                __shared__ nd4j::random::RandomBuffer *buffer;
                __shared__ unsigned char *cB;
                __shared__ unsigned char *dB;
                __shared__ nd4j::random::RandomBuffer *devBuffer;
                if (threadIdx.x == 0) {
                    extern __shared__ unsigned char shmem[];
                    buffer = (nd4j::random::RandomBuffer *) shmem;
                    cB = shmem;
                    devBuffer = reinterpret_cast<nd4j::random::RandomBuffer *> (state);
                    dB = reinterpret_cast<unsigned char *> (state);
                }
                __syncthreads();

                // using this loop instead of memcpy
                for (int e = threadIdx.x; e < sizeof(nd4j::random::RandomBuffer); e+= blockDim.x) {
                    cB[e] = dB[e];
                }
                __syncthreads();

                int tid = blockIdx.x * blockDim.x + threadIdx.x;

                if (ews >= 1) {
                    for (Nd4jLong x = tid; x < length; x += blockDim.x * gridDim.x) {
                        z[x * ews] = OpClass::op(x, length, buffer, extraArguments);
                    }
                } else {
                    // ind2sub branch
                    Nd4jLong zCoord[MAX_RANK];

                    int zRank = shape::rank(zShapeBuffer);
                    auto zShape = shape::shapeOf(zShapeBuffer);
                    auto zStride = shape::stride(zShapeBuffer);

                    for (Nd4jLong i = tid; i < length; i += blockDim.x * gridDim.x) {
                        shape::ind2sub(zRank, zShape, i, length, zCoord);

                        auto zOffset2 = shape::getOffset(0, zShape, zStride, zCoord, zRank);

                        z[zOffset2] = OpClass::op(i, length, buffer,  extraArguments);
                    }
                }

                __syncthreads();
                devBuffer->rewind(length);
            }

        template <>
        _CUDA_H void RandomFunction<float>::executeCudaSingle(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, float *z, Nd4jLong *zShapeBuffer, float *extraArguments) {
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (stateHost);
            Nd4jPointer state = buffer->getDevicePointer();

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, float, PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float16>::executeCudaSingle(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, float16 *z, Nd4jLong *zShapeBuffer, float16 *extraArguments) {
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (stateHost);
            Nd4jPointer state = buffer->getDevicePointer();

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, float16, PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<double>::executeCudaSingle(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, double *z, Nd4jLong *zShapeBuffer, double *extraArguments) {
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (stateHost);
            Nd4jPointer state = buffer->getDevicePointer();

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomSingle, double, PARAMS(state, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float>::executeCudaDouble(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, float *x, Nd4jLong *xShapeBuffer, float *z, Nd4jLong *zShapeBuffer, float *extraArguments) {
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (stateHost);
            Nd4jPointer state = buffer->getDevicePointer();

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, float, PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }


        template <>
        _CUDA_H void RandomFunction<float16>::executeCudaDouble(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, float16 *x, Nd4jLong *xShapeBuffer, float16 *z, Nd4jLong *zShapeBuffer, float16 *extraArguments) {
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (stateHost);
            Nd4jPointer state = buffer->getDevicePointer();

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, float16, PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<double>::executeCudaDouble(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, double *x, Nd4jLong *xShapeBuffer, double *z, Nd4jLong *zShapeBuffer, double *extraArguments) {
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (stateHost);
            Nd4jPointer state = buffer->getDevicePointer();

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomDouble, double, PARAMS(state, x, xShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float>::executeCudaTriple(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, float *x, Nd4jLong *xShapeBuffer, float *y, Nd4jLong *yShapeBuffer, float *z, Nd4jLong *zShapeBuffer, float *extraArguments) {
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (stateHost);
            Nd4jPointer state = buffer->getDevicePointer();

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, float, PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }

        template <>
        _CUDA_H void RandomFunction<float16>::executeCudaTriple(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, float16 *x, Nd4jLong *xShapeBuffer, float16 *y, Nd4jLong *yShapeBuffer, float16 *z, Nd4jLong *zShapeBuffer, float16 *extraArguments) {
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (stateHost);
            Nd4jPointer state = buffer->getDevicePointer();

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, float16, PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }



        template <>
        _CUDA_H void RandomFunction<double>::executeCudaTriple(dim3& launchDims, Nd4jPointer *extraPointers, int opNum, Nd4jPointer stateHost, double *x, Nd4jLong *xShapeBuffer, double *y, Nd4jLong *yShapeBuffer, double *z, Nd4jLong *zShapeBuffer, double *extraArguments) {
            hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

            nd4j::random::RandomBuffer *buffer = reinterpret_cast<nd4j::random::RandomBuffer *> (stateHost);
            Nd4jPointer state = buffer->getDevicePointer();

            // this macro builds bunch of IF/ELSE selectors for kernel launch
            DISPATCH_SIMPLE(randomTriple, double, PARAMS(state, x, xShapeBuffer, y, yShapeBuffer, z, zShapeBuffer, extraArguments), OPS_A(RANDOM_OPS))

            DEBUG_KERNEL(stream, opNum);
        }



        template class ND4J_EXPORT RandomFunction<float>;
        template class ND4J_EXPORT RandomFunction<float16>;
        template class ND4J_EXPORT RandomFunction<double>;
    }
}
